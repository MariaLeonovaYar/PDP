#include "hip/hip_runtime.h"
﻿#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

__global__ void calculateK(float* k1, float* k2, float* k3, float* k4, float a, float b, float xn, float yn, float h, int i) {
    k1[i] = h * cos(a * xn) - b * yn;
    k2[i] = h * (cos(a * (xn + h / 2.0)) - b * (yn + k1[i] * h / 2.0));
    k3[i] = h * (cos(a * (xn + h / 2.0)) - b * (yn + k2[i] * h / 2.0));
    k4[i] = h * (cos(a * (xn + h)) - b * (yn + k3[i] * h));
}

__global__ void calculateRez(float* yn, float* k1, float* k2, float* k3, float* k4, float h, int i) {
    yn[i] = yn[i] + (k1[i] + 2 * k2[i] + 2 * k3[i] + k4[i]) / 6.0;
}

int main() {

    srand(NULL);

    float x0 = 0, h = 0.0002, y0 = 0.8;
    float xn = x0;
    int finish = 10000, m = 100;

    float* k1, * k2, * k3, * k4, * yn;
    float* c_k1, * c_k2, * c_k3, * c_k4, * c_yn;

    k1 = (float*)malloc(m * sizeof(float));
    k2 = (float*)malloc(m * sizeof(float));
    k3 = (float*)malloc(m * sizeof(float));
    k4 = (float*)malloc(m * sizeof(float));
    yn = (float*)malloc(m * sizeof(float));

    for (int i = 0; i < finish; i++) {

        float a = rand() % 1000;
        float b = rand() % 1000;

        hipMalloc((void**)&c_k1, m * sizeof(float));
        hipMalloc((void**)&c_k2, m * sizeof(float));
        hipMalloc((void**)&c_k3, m * sizeof(float));
        hipMalloc((void**)&c_k4, m * sizeof(float));
        hipMalloc((void**)&c_yn, m * sizeof(float));

        hipMemcpy(k1, c_k1, m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(k2, c_k2, m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(k3, c_k3, m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(k4, c_k4, m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(yn, c_yn, m * sizeof(float), hipMemcpyHostToDevice);

        calculateK << <1, m >> > (c_k1, c_k2, c_k3, c_k4, a, b, xn, y0, h, i);
        calculateRez << <1, m >> > (c_yn, c_k1, c_k2, c_k3, c_k4, h, i);

        hipMemcpy(c_k1, k1, m * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(c_k2, k2, m * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(c_k3, k3, m * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(c_k4, k4, m * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(c_yn, yn, m * sizeof(float), hipMemcpyDeviceToHost);

        for (int i = 0; i < finish; i++) {
           cout << yn[i] << " ";
        }

        cout << "\n";

        xn += h;
    }

    free(k1);
    free(k2);
    free(k3);
    free(k4);
    free(yn);

    hipFree(c_k1);
    hipFree(c_k2);
    hipFree(c_k3);
    hipFree(c_k4);
    hipFree(c_yn);


    return 0;
}